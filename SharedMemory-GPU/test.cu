
#include <hip/hip_runtime.h>
#include <iostream>
#define imin(a,b) (a<b?a:b)

const int N = 1000000000 * 33; 
const int threadsPerBlock = 256; 
const int blocksPerGrid = imin(32, (N+threadsPerBlock-1) / threadsPerBlock);

__global__ void dot(float *a, float *b, float *c) {
    __shared__ float cache[threadsPerBlock];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;
    float temp = 0;
    
    while (tid < N) {
        temp += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }
    
    // set the cache values
    cache[cacheIndex] = temp;
    
    // synchronize threads in this block
    __syncthreads();
    
    // for reductions, threadsPerBlock must be a power of 2
    // because of the following code
    int i = blockDim.x/2;
    while (i != 0) {
        if (cacheIndex < i)
            cache[cacheIndex] += cache[cacheIndex + i];
        __syncthreads();
        i /= 2;
    }
    
    if (cacheIndex == 0)
        c[blockIdx.x] = cache[0]; 
}

int main(void) {
    float *a, *b, c, *partial_c;
    float *dev_a, *dev_b, *dev_partial_c;
    
    // allocate memory on the CPU side
    a = (float*)malloc(N*sizeof(float));
    b = (float*)malloc(N*sizeof(float));
    partial_c = (float*)malloc(blocksPerGrid*sizeof(float));
    
    // allocate the memory on the GPU
    hipMalloc((void**)&dev_a, N*sizeof(float));
    hipMalloc((void**)&dev_b, N*sizeof(float));
    hipMalloc((void**)&dev_partial_c, blocksPerGrid*sizeof(float));
    
    // fill in the host memory with data
    for (int i=0; i<N; i++) {
        a[i] = i;
        b[i] = i;
    }
    
    // copy the arrays 'a' and 'b' to the GPU
    hipMemcpy(dev_a, a, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N*sizeof(float), hipMemcpyHostToDevice);
    
    dot<<<blocksPerGrid,threadsPerBlock>>>(dev_a, dev_b, dev_partial_c);
    
    // copy the array 'c' back from the GPU to the CPU
    hipMemcpy(partial_c, dev_partial_c, blocksPerGrid*sizeof(float), hipMemcpyDeviceToHost);
    
    // finish up on the CPU side
    c = 0;
    for (int i=0; i<blocksPerGrid; i++) {
        c += partial_c[i];
    }
    
    #define sum_squares(x) (x*(x+1)*(2*x+1)/6)
    printf("Result %f\n", c);
    
    // free memory on the GPU side
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_partial_c);
    
    // free memory on the CPU side
    free(a);
    free(b);
    free(partial_c);
    
    return 0;
}