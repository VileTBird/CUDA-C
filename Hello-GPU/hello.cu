
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void helloFromGPU()
{
	printf("Hello dude!\n");
}

int main()
{
	helloFromGPU<<<1, 10>>>();
	hipDeviceSynchronize();
	return 0;
}
