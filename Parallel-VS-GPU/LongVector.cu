
#include <hip/hip_runtime.h>
#include <iostream>

# define N 10000

__global__ void add(int *a, int *b, int *c)
{

    // so basically this formula makes it so that each id across each thread in each fucking block sequential and incremental
    // how? i have no fucking clue how it fucking works but it just does i even did it on pen & paper to figure out intuiton
    // god in each block if we have 4 threads itll repeat with 0, 1, 2, 3 for threadIdx.x
    // if we add the block number lets say we have 4 blocks, then it would be 0 + 0, 1+0... 0 + 1, 1 + 1, do u see how it can create overlap
    // here when we multiply it by the number of threads which is blockDim with our block id
    // then 0 + 0 * 4, 1+ 0 * 4... 3 + 0 * 4, 0 + 1 * 4, 1 + 1 * 4, do u see how it makes it incremental?
    // so yeah its cool but i dont get it just that it works ill just remember it for later
    int id = threadIdx.x + blockIdx.x * blockDim.x;

    if(id < N)
    {
        c[id] = a[id] + b[id];
    }
}

int main(void)
{
    int a[N], b[N], c[N];
    int *device_a, *device_b, *device_c;

    hipMalloc((void**)&device_a, N * sizeof(int));
    hipMalloc((void**)&device_b, N * sizeof(int));
    hipMalloc((void**)&device_c, N * sizeof(int));

    for(int i = 0; i < N; i++)
    {
        a[i] = i;
        b[i] = -i;
    }

    hipMemcpy(device_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_b, b, N * sizeof(int), hipMemcpyHostToDevice);

    add<<<(N + 127)/128,128>>>(device_a, device_b, device_c);

    hipMemcpy(c, device_c, N * sizeof(int), hipMemcpyDeviceToHost);

    for(int i = 0; i < N; i++)
    {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    hipFree(device_a);
    hipFree(device_b);
    hipFree(device_c);
    return 0;
}