
#include <hip/hip_runtime.h>
# include <iostream>
# define N 1000

__global__ void _vdot(int *a, int *b, int *c)
{
	__shared__ int cache[128];

	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	int temp = 0;

	int cacheID = threadIdx.x;
	while(tid < N)
	{
		temp += a[tid] * b[tid];
		tid += blockDim.x * gridDim.x;
	}

	__syncthreads();

	cache[cacheID] = temp;

	int i = blockDim.x / 2;

	while(i != 0)
	{
		if(cacheID < i)
		{
			cache[cacheID] += cache[cacheID + i];
		}
		__syncthreads();
		i /= 2;
	}

	if (cacheID == 0)
	{
		c[blockIdx.x] = cache[cacheID];
	}
}

int main(void)
{
	int a[N], b[N], c[(N+127)/128];

	int *d_a, *d_b, *d_c;

	size_t size = N * sizeof(int);
	for(int i = 0; i < N; i++)
	{
		a[i] = i;

		b[i] = i;
	}
	
	hipMalloc((void**)&d_a, size);
	hipMalloc((void**)&d_b, size);
	hipMalloc((void**)&d_c, ((N+127)/128) * sizeof(int));

	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	_vdot<<<(N+127)/128, 128>>>(d_a, d_b, d_c);

	hipMemcpy(c, d_c, ((N+127)/128) * sizeof(int), hipMemcpyDeviceToHost);

	int res = 0;
	for(int i = 0; i < ((N+127)/128) ; i++)
	{
		res += c[i];
	}
	
	printf("result: %d\n", res);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	return 0;
}
