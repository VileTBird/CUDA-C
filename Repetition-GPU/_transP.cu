
#include <hip/hip_runtime.h>
# include <iostream>
# define N 10 

__global__ void transposeGPU(int *a, int *c, int columns, int rows)
{
    int rowId = threadIdx.x + blockIdx.x * blockDim.x;
    int columnId = threadIdx.y + blockIdx.y * blockDim.y;

    while(rowId < columns && columnId < columns)
    {
        c[rowId * columns + columnId] = a[columnId * rows + rowId];

        rowId += blockDim.x * gridDim.x;
        columnId += blockDim.y * gridDim.y;
    }
}

void printMatrix(int *matrix, int columns)
{   
    for(int i = 0; i < N; i++)
    {
        for(int j = 0; j < N; j++)
        {
            printf(" %d ", matrix[i * columns + j]);
        }
        printf("\n");
    }
    printf("\n");
}

void transposeCPU(int *a, int *b, int columns, int rows)
{
    for(int i = 0; i < N; i++)
    {
        for(int j = 0; j < N; j++)
        {
            b[i * columns + j] = a[j * rows + i];
        }
    }
}

int main(void)
{
    size_t size = N * N * sizeof(int);

    int *a, *b;

    a = (int*) malloc(size);
    b = (int*) malloc(size);

    for(int i = 0; i < N * N; i++)
    {
        a[i] = i;
    }

    transposeCPU(a, b, N, N);

    printMatrix(a, N);
    printMatrix(b, N);

    int *c;
    int *d_a, *d_c;
    c = (int*) malloc(size);

    hipMalloc((void**)&d_c, size);
    hipMalloc((void**)&d_a, size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    
    dim3 blockDim(12, 12, 1);

    int blockCount = 0;
    if (N > blockDim.x)
    {
        blockCount = (N+11)/12;
    }
    else
    {
        blockCount = 32;
    }
    dim3 gridDim(blockCount, blockCount, 1);
    transposeGPU<<<gridDim, blockDim>>>(d_a, d_c, N, N);

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    printMatrix(c, N);
    hipFree(d_c);
    hipFree(d_a);

    free(a);
    free(b);
    free(c);

    return 0;
}