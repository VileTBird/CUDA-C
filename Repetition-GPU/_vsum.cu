
#include <hip/hip_runtime.h>
#include <iostream>
#define N 10

__global__ void v_sum(int *A, int *B, int *C)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    while(tid < N)
    {
        C[tid] = A[tid] + B[tid];
        tid += blockDim.x * gridDim.x;
    }
}

int main(void)
{

    size_t size = N * sizeof(int);
    int A[N], B[N], C[N];

    int *d_A, *d_B, *d_C;

    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    for(int i = 0; i < N; i++)
    {
        A[i] = i;
        B[i] = -i;
    }

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
    

    v_sum<<<32, 256>>>(d_A, d_B, d_C);

    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    for(int i = 0; i < N; i++)
    {
        printf("%d + %d = %d\n", A[i], B[i], C[i]);
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);


    return 0;
}