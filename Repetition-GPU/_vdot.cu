
#include <hip/hip_runtime.h>
#include <iostream>
#define N 1000

__global__ void _vDot(float *a, float *b, float *c)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    while(tid < N)
    {
        c[tid] = a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }
}

int main(void)
{
    float a[N], b[N], c[N];

    float *d_a, *d_b, *d_c;

    size_t size = N * sizeof(float);
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    for(int i = 0; i < N; i++)
    {
        a[i] = i;
        b[i] = i;
    }

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, a, size, hipMemcpyHostToDevice);

    _vDot<<<32, 256>>>(d_a, d_b, d_c);

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    float res = 0;
    for(int i = 0; i < N; i++)
    {
        printf("%f * %f", a[i], b[i]);
        printf(" +");
        res += c[i];
    }

    printf(" =%f\n", res);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}