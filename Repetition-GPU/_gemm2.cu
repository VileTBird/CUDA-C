
#include <hip/hip_runtime.h>
#include <iostream>

# define N 2

__global__ void transposeGPU(int *a, int *c, int columns, int rows)
{
    int rowId = threadIdx.x + blockIdx.x * blockDim.x;
    int columnId = threadIdx.y + blockIdx.y * blockDim.y;

    while(rowId < columns && columnId < columns)
    {
        c[rowId * columns + columnId] = a[columnId * rows + rowId];

        rowId += blockDim.x * gridDim.x;
        columnId += blockDim.y * gridDim.y;
    }
}

void printMatrix(int *matrix, int columns)
{   
    for(int i = 0; i < N; i++)
    {
        for(int j = 0; j < N; j++)
        {
            printf(" %d ", matrix[i * columns + j]);
        }
        printf("\n");
    }
    printf("\n");
}

void gemmCPU(int *a, int *b, int *c, int columns, int rows)
{
    for(int rowId = 0; rowId < N; rowId++)
    {
        for(int columnId = 0; columnId < N; columnId++)
        {
            for(int elementId = 0; elementId < N; elementId++)
            {
                // i donmt really get this clusterfuck but im sure if i can visualize this i can write the kernel myself without outside refernce
                // lol this was just throwing darts at a wall until i hit th eright combination although i understand the memory layout its still fucked
                // very confusing
                c[rowId * columns + columnId] += a[rowId * columns + elementId] * b[elementId * rows + columnId]; 
            }
        }
    }
}

int main(void)
{
    size_t size = N * N * sizeof(int);

    int *a, *b, *d;

    a = (int*) malloc(size);
    b = (int*) malloc(size);

    d = (int*) malloc(size);

    for(int i = 0; i < N * N; i++)
    {
        a[i] = i;
        b[i] = i + 1;
    }

    gemmCPU(a, b, d, N, N);

    printMatrix(a, N);
    printMatrix(b, N);

    printMatrix(d, N);

    /*
    int *c;
    int *d_a, *d_c;
    c = (int*) malloc(size);

    cudaMalloc((void**)&d_c, size);
    cudaMalloc((void**)&d_a, size);

    cudaMemcpy(d_a, a, size, cudaMemcpyHostToDevice);
    
    dim3 blockDim(12, 12, 1);

    int blockCount = 0;
    if (N > blockDim.x)
    {
        blockCount = (N+11)/12;
    }
    else
    {
        blockCount = 32;
    }
    dim3 gridDim(blockCount, blockCount, 1);
    transposeGPU<<<gridDim, blockDim>>>(d_a, d_c, N, N);

    cudaMemcpy(c, d_c, size, cudaMemcpyDeviceToHost);

    printMatrix(c, N);
    cudaFree(d_c);
    cudaFree(d_a);

    free(a);
    free(b);
    free(c);

    return 0;
    */
}