
#include <hip/hip_runtime.h>
#include <iostream>
# define Count 2    

__global__ void gemmGPU(int *a, int *b, int *c, int M, int N, int K)
{
    int colId = threadIdx.y + blockIdx.y * blockDim.y;
    int rowId = threadIdx.x + blockIdx.x * blockDim.x;

    while(colId < N && rowId < M)
    {
        for(int k = 0; k < K; k++)
        {
            c[rowId * N + colId] += a[rowId * K + k] * b[k * N + colId];
        }

        colId += blockDim.y * gridDim.y;
        rowId += blockDim.x * gridDim.x;
    }
}

void printMatrix(int *a, int M, int N)
{
    for(int i = 0; i < M; i++)
    {
        for(int j = 0; j < N; j++)
        {
            printf(" %d ", a[i * N + j]);
        }
        printf("\n");
    }
    printf("\n");
}

void gemmCPU(int *a, int *b, int *c, int M, int N, int K)
{
    for(int i = 0; i < M; i++)
    {
        for(int j = 0; j < N; j++)
        {
            for(int k = 0; k < K; k++)
            {
                // N * k cus well total number of columns = total number of elements in a row
                // when u multiply it with k itll be shifting to each corresponding element in that column
                // when u add j to it itll offset column by 1

                // i * K basically gets the same as well it gets the row but i is slow to incrmeent
                // i wont incrmeent until i hits max, in other words adding k in eachiteration
                // will add each element in a row before i increments and takes it to thenext row
                c[i * N + j] += a[i * K + k] * b[k * N + j];
            }
        }
    }
}

int main(void)
{

    int *a, *b, *c, *d;
    size_t size = Count * Count * sizeof(int);
    a = (int*) malloc(size);
    b = (int*) malloc(size);
    c = (int*) malloc(size);
    d = (int*) malloc(size);

    for(int i = 0; i < Count * Count; i++)
    {
        a[i] = i;
        b[i] = i;
    }

    // just in case our values wont get stacked on garbage values
    memset(c, 0, size);
    memset(d, 0, size);

    gemmCPU(a, b, c, Count, Count, Count);

    printMatrix(a, Count, Count);
    printMatrix(b, Count, Count);
    printMatrix(c, Count, Count);

    dim3 gridDim((Count+11)/12, (Count+11)/12);
    dim3 blockDim(12, 12);

    int *d_a, *d_b, *d_c;

    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_c, size);
    hipMalloc((void**)&d_b, size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    gemmGPU<<<gridDim, blockDim>>>(d_a, d_b, d_c, Count, Count, Count);

    hipMemcpy(d, d_c, size, hipMemcpyDeviceToHost);

    printMatrix(d, Count, Count);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(a);
    free(b);
    free(c);
    free(d);
    return 0;
}