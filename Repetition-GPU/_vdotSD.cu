
#include <hip/hip_runtime.h>
#include <iostream>
#define N 1000

__global__ void _vDot(float *a, float *b, float *c)
{

    __shared__ float cache[256];

    int cacheId = threadIdx.x;
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    double temp = 0;
    while(tid < N)
    {
        temp += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }

    __syncthreads();
    cache[cacheId] = temp;

    int i = blockDim.x / 2;

    while(i != 0)
    {
        if(cacheId < i)
        {
            cache[cacheId] += cache[cacheId + i];
        }
        __syncthreads();
        i = i/2;
    }

    if(cacheId == 0)
    {
        c[blockIdx.x] = cache[0];
    }
}

int main(void)
{
    float a[N], b[N], c[N];

    float *d_a, *d_b, *d_c;

    size_t size = N * sizeof(float);

    size_t csize = (N+255)/256 * sizeof(float);
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, csize);

    for(int i = 0; i < N; i++)
    {
        a[i] = i;
        b[i] = i;
    }

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    _vDot<<<(N+255)/256, 256>>>(d_a, d_b, d_c);

    hipMemcpy(c, d_c, csize, hipMemcpyDeviceToHost);

    double res = 0;
    for(int i = 0; i < (N+255)/256; i++)
    {
        res += c[i];
    }

    printf(" =%f\n", res);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}