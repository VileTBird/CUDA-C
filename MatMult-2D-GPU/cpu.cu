
#include <hip/hip_runtime.h>
#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <functional>
#include <iostream>
#include <vector>
# define N 10
#include <iomanip>

__global__ void d_transpose(int *d_a, int *d_c, int rows, int columns)
{
    // here even if we flip it, shit doesnt matter cus at the end of the day all we need is
    // every single combination of row id and column id for atleast one thread to do thecorresponding calculation for an accurate transpose.
    int columnId = threadIdx.x + blockDim.x * blockIdx.x;
    int rowId = threadIdx.y + blockDim.y * blockIdx.y;

    if(rowId < rows && columnId < columns)
    {
        d_c[rowId * columns + columnId] = d_a[columnId * rows + rowId];
        __syncthreads();
    }
}

void print_matrix(const int *matrix, int rows, int cols, const std::string& name) {
    std::cout << name << ":\n";
    for (int row = 0; row < rows; row++) {
        for (int col = 0; col < cols; col++) {
            // i think this is pretty simple too tbh, i could write this myself next time for repeittion tmrw
            // all it is doing is printing each element in row major order sequentially with spaces and line breaks
            std::cout << std::setw(4) << matrix[row * cols + col] << " ";
        }
        std::cout << "\n";
    }
    std::cout << std::endl;
}

void transpose(int *A, int *B, int rows, int columns)
{
    for(int rowId = 0; rowId < rows; rowId++ )
    {
        for(int columnId = 0; columnId < columns; columnId++)
        {
            // here we're basically saying that each column-major id for our matrix would be the 
            // main matrix's row major id, which in other words flips i, j to j, i correspondingly
            // the formula in itself is pretty smart when it comes to flattening a multidimensional matrix into 1d 
            // baically in row major order or rowId * column + columnId
            // we basically say jump to the row our element is at with rowId * columns, cus columns = number of elements in a row
            // when u multiply that with the column id u jump across that many columns to our column, once u jump there when u add
            // col id to the mix u basically jump to the id of our element within that row, thats basically it.
            B[columnId * rows + rowId] = A[rowId * columns + columnId]; 
        }
    }
}
int main(void)
{
    int *h_a = (int*)malloc(N * N * sizeof(int));
    int *h_b = (int*)malloc(N * N * sizeof(int));

    int *h_c = (int*)malloc(N * N * sizeof(int));

    int *d_a, *d_c;

    for(int i = 0; i < N * N; i++)
    {
        h_a[i] = rand() % 100;
    }
    print_matrix(h_a, N, N, "Original Matrix A");
    transpose(h_a, h_b, N, N);

    print_matrix(h_b, N, N, "Transposed Matrix B");

    hipMalloc((void**)&d_a, N * N * sizeof(int));
    hipMalloc((void**)&d_c, N * N * sizeof(int));

    hipMemcpy(d_a, h_a, N * N * sizeof(int), hipMemcpyHostToDevice);

    // initially my blockDim was kinda problematic with N, N, 0
    // Tomorrow we have to figure out how to break this thing, what if row, col weretn equal waht if it were greater what if it was lower eh and eh
    // figure out how stride loop would work for the kernel and so on, now im tired.
    dim3 blockDim(N, N, 1);
    d_transpose<<<32, blockDim>>>(d_a, d_c, N, N);

    hipMemcpy(h_c, d_c, N * N * sizeof(int), hipMemcpyDeviceToHost);

    print_matrix(h_c, N, N, "Transposed Matrix C");

    hipFree(d_a);
    hipFree(d_c);
    

    free(h_a);
    free(h_b);
    

    return 0;
}