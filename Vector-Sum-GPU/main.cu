
#include <hip/hip_runtime.h>
#include <iostream>

// define kernel

# define N 10

__global__ void add(int *a, int *b, int *c)
{
    int tid = blockIdx.x;

    if (tid < N)
    {
        c[tid] = a[tid] + b[tid];
    }
}

int main(void)
{
    int a[N], b[N], c[N];
    int *device_a, *device_b, *device_c;

    hipMalloc( (void**)&device_a, N * sizeof(int) );
    hipMalloc( (void**)&device_b, N * sizeof(int) );
    hipMalloc( (void**)&device_c, N * sizeof(int) );
    for (int i = 0; i < N; i++)
    {
        a[i] = i;
        b[i] = i;
    }

    hipMemcpy(device_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_b, b, N * sizeof(int), hipMemcpyHostToDevice);
    add<<<10, 1>>>(device_a, device_b, device_c);

    hipMemcpy(c, device_c, N * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(device_a);
    hipFree(device_b);
    hipFree(device_c);

    int i = 0;
    while(i < N)
    {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
        i += 1;
    }
    return 0;
}