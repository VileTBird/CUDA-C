
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

void gemmCPU(int *a, int *b, int *c, int m, int k, int n)
{
    for (int i = 0; i < m; i++)         // rows of A
    {
        for (int j = 0; j < n; j++)     // columns of B
        {
            c[i * n + j] = 0;           // initialize result cell
            for (int x = 0; x < k; x++) // shared dim: cols of A, rows of B
            {
                c[i * n + j] += a[i * k + x] * b[x * n + j];
            }
        }
    }
}

void printMatrixWithLabel(const char* label, int *a, int cols, int rows)
{
    printf("%s (%dx%d):\n", label, rows, cols);
    for (int i = 0; i < rows; i++)
    {
        for (int j = 0; j < cols; j++)
        {
            printf("%4d ", a[i * cols + j]);
        }
        printf("\n");
    }
    printf("\n");
}

int main(void)
{
    int m = 2, k = 3, n = 2;

    // Matrix A (2x3)
    int A[] = {
        1, 2, 3,
        4, 5, 6
    };

    // Matrix B (3x2)
    int B[] = {
        1, 4,
        3, 6,
        5, 8
    };

    // Output matrix C (2x2)
    int* C = (int*)malloc(m * n * sizeof(int));

    printMatrixWithLabel("Matrix A", A, k, m);  // A is m x k
    printMatrixWithLabel("Matrix B", B, n, k);  // B is k x n

    gemmCPU(A, B, C, m, k, n);

    printMatrixWithLabel("Result (A x B)", C, n, m);  // C is m x n

    free(C);
    return 0;
}
