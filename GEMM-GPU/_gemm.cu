
#include <hip/hip_runtime.h>
# include <iostream>
# define count 2


__global__ void gemmGPU(int *a, int *b, int *c, int M, int N, int K)
{
    int colId = threadIdx.x + blockIdx.x * blockDim.x;
    int rowId = threadIdx.y + blockIdx.y * blockDim.y;

    while(rowId < M && colId < N)
    {
        for(int k = 0; k < K; k++)
        {
            c[rowId * N + colId] += a[rowId * K + k] * b[k * N + colId];
        }
        rowId += blockDim.y * gridDim.y;

        colId += blockDim.x * gridDim.x;
    }
}
void printMatrix(int *a, int M, int N)
{
    for(int i = 0; i < M; i++){
        for(int j = 0; j < N; j++)
        {
            printf(" %d ", a[i * N + j]);
        }
        printf("\n");
    }
    printf("\n");
}

void gemmCPU(int *a, int *b, int *c, int M, int N, int K)
{
    for(int i = 0; i < M; i++)
    {
        for(int j = 0; j < N; j++)
        {
            for(int k = 0; k < K; k++)
            {
                c[i * N + j] += a[i * K + k] * b[k * N + j];
            }
        }
    }
}

int main(void)
{
    int *a, *b, *c;

    size_t size = count * count * sizeof(int);

    a = (int*) malloc(size);
    b = (int*) malloc(size);
    c = (int*) malloc(size);

    for (int i = 0;  i < count * count; i++)
    {
        a[i] = i;
        b[i] = i;
    }

    gemmCPU(a, b, c, count, count, count);

    printMatrix(a, count, count);
    printMatrix(b, count, count);
    printMatrix(c, count, count);

    int *d = (int*) malloc(size);

    int *d_a, *d_b, *d_c;

    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    dim3 blockDim(16, 16);
    dim3 gridDim(count+15/16, count+15/16);
    gemmGPU<<<gridDim, blockDim>>>(a, b, d, count, count, count);

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    printMatrix(d, count, count);
    free(a);
    free(b);
    free(c);
    free(d);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;

}