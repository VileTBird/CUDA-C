
#include <hip/hip_runtime.h>
# include <iostream>
# define N 2

void printMatrix(int *a, int m, int n)
{
    for(int i = 0; i < m; i++)
    {
        for(int j = 0; j < n; j++)
        {
            printf(" %d ", a[i * n + j]);
        }
        printf("\n");
    }
    printf("\n");
}

void gemmCPU(int *a, int *b, int *c, int m, int n, int K)
{
    // m x k -> dimension of first matrix a
    // k x n -> dimension of second matrix b
    // m x n -> dimension of resulting matrix c

    // for each row in matrix a
    for(int i = 0; i < m; i++)
    {
        // for each column in matrix b
        for(int j = 0; j < n; j++)
        {
            // for each element in both row and column of matrix a, b
            for(int k = 0; k < K; k++)
            {
                // for each element i, j in matrix c, for each element k 
                // in row i * K of matrix a, we multiply with each element j
                // in each row k * N of matrix b. 
                // in other words one we iteratue thru each element in row of matrix 1
                // by offsetting each elemnt by k for each row i * k 
                // simultaneously we use row major order with k * n, to jump to
                // each row and get the j corresponding element of matrix b which would give us columns
                c[i * n + j] += a[i * K + k] * b[k * n + j];
            }
        }
    }
}

int main(void)
{
    int *a, *b, *c;

    size_t size = N * N * sizeof(int);

    a = (int*) malloc(size);
    b = (int*) malloc(size);
    c = (int*) malloc(size);

    for(int i = 0; i < N * N; i++)
    {
        a[i] = i;
        b[i] = i;
    }

    printMatrix(a, N, N);

    printMatrix(b, N, N);

    gemmCPU(a, b, c, N, N, N);

    printMatrix(c, N, N);
    free(a);
    free(b);
    free(c);
    return 0;
}